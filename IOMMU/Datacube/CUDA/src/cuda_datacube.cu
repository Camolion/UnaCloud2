#include "hip/hip_runtime.h"
#include "cuda_datacube.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#define MEDIAN(array, size, type) (size % 2 == 0 ? (type)((array[size/2]+array[(size/2)-1])/2) : array[(int)floor(size/2.0)]);
#define CUDA_MALLOC(array, arraySize, type) hipMalloc((void **)&array, sizeof(type)*arraySize);
#define CUDA_MALLOC_HOST(array, arraySize, type) hipHostMalloc((void **)&array, sizeof(type)*arraySize);
#define NBANDS 6

/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

__global__ void applyMasks16Kernel(short *images, bool *cmasks, int width, int height, int bands, short nullValue){

	int x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y = threadIdx.y + (blockDim.y * blockIdx.y);
	int z = threadIdx.z + (blockDim.z * blockIdx.z);

	int idx = x + (y * width) + (z * height * width);

	if(x < width && y < height && z < bands){
		if(!cmasks[idx])
			images[idx] = nullValue;
	}

}
__global__ void normImages16Kernel(short *images, double *means, double meanMeans, double *stdDevs, double meanStdDevs, int width, int height, int  bands, short nullValue){

	int x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y = threadIdx.y + (blockDim.y * blockIdx.y);
	int z = threadIdx.z + (blockDim.z * blockIdx.z);

	int idx = x + (y * width) + (z * height * width);

	if(x < width && y < height && z < bands){
		if(images[idx] != nullValue)
			images[idx] = (((double)images[idx]-means[z])/stdDevs[z])*meanStdDevs+meanMeans;
	}
}

__global__ void dcMediansBandInt16Kernel(short *images, short *mediansBand, int width, int height, int bands, int minValidCount, short nullValue){

	int x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y = threadIdx.y + (blockDim.y * blockIdx.y);

	int idx = x + (y * width);
	int imgSize = width*height;

	int i, j, bandArrIdx;

	if(x < width && y < height){

		int validCounter = 0;
		int pixel[NBANDS];
		for(i = 0; i < bands; i++){
			bandArrIdx = idx + (imgSize*i);
			if(images[bandArrIdx] != nullValue){
				pixel[validCounter] = images[bandArrIdx];
				validCounter++;
			}
		}

		if(minValidCount <= validCounter){
			for(i = 0; i < validCounter-1; i++){
				for(j = i+1; j < validCounter; j++){
					if(pixel[i] > pixel[j]){
						short tmp = pixel[i];
						pixel[i] = pixel[j];
						pixel[j] = tmp;
					}
				}
			}
			mediansBand[idx] = MEDIAN(pixel, validCounter, short);
		}else{
			mediansBand[idx] = nullValue;
		}
	}
}

extern "C" {
	void dcMediansBandInt16(short *src_images, short *medianc, bool *cmask, int width, int height, int bands, bool normalized, int minValidCount, short nullValue){
	
		int i = 0, j = 0;
		int arrSize = width * height * bands;
		int imgSize = width * height;
		int xblocks, yblocks, zblocks;

		// Create images array on the CUDA device
		short *d_images;
		CUDA_MALLOC(d_images, arrSize, short);
		checkCUDAError("Error in CUDA Malloc for images");
		hipMemcpy(d_images, src_images, sizeof(short)*arrSize, hipMemcpyHostToDevice);
		checkCUDAError("Error in CUDA Memcpy for images");

		// Creates Cloud mask on the CUDA device
		bool *d_cmask;
		CUDA_MALLOC(d_cmask, arrSize, bool);
		checkCUDAError("Error in CUDA Malloc for cloud masks");	
		hipMemcpy(d_cmask, cmask, sizeof(bool)*arrSize, hipMemcpyHostToDevice);
		checkCUDAError("Error in CUDA Malloc for cloud masks");

		// Creates medians compound on the CUDA device
		short *d_medianc;
		CUDA_MALLOC(d_medianc, imgSize, short);
		checkCUDAError("Error in CUDA Malloc for Medians Band");

		// Apply Masks
		xblocks = ceil(width/8.0);
		yblocks = ceil(height/8.0);
		zblocks = ceil(bands/8.0);

		dim3 maskBlocks(xblocks, yblocks, zblocks);
		dim3 maskThreads(8,8,8);

		applyMasks16Kernel<<<maskBlocks, maskThreads>>>(d_images, d_cmask, width, height, bands, nullValue);
		hipDeviceSynchronize();
		checkCUDAError("Error in Apply Masks");

		hipFree(d_cmask);

		if(normalized){

			short *images;
			CUDA_MALLOC_HOST(images, arrSize, short);
			hipMemcpy(images, d_images, sizeof(short)*arrSize, hipMemcpyDeviceToHost);

			double means[bands];
			double stddev[bands];
			double meanMeans = 0.0;
			double meanStddev = 0.0;
			int validCounter, idx;

			// Mean for each band
			for(i = 0; i < bands; i++){
				validCounter = 0;
				means[i] = 0.0;
				for(j = 0; j < imgSize; j++){
					idx = j+(imgSize*i);
					if(images[idx] != nullValue){
						means[i] += images[idx];
						validCounter++;
					}
				}
				if(validCounter > 0)
					means[i] /= validCounter; 
				//printf("Means %d: %f\n", i, means[i]);
			}
	
			// Standard deviation for each band
			for(i = 0; i < bands; i++){
				validCounter = 0;
				stddev[i] = 0.0;
				for(j = 0; j < imgSize; j++){
					idx = j+(imgSize*i);
					if(images[idx] != nullValue){
						stddev[i] += pow((images[idx] - means[i]), 2);
						validCounter++;
					}
				}
				if(validCounter > 0)
					stddev[i] = sqrt(stddev[i] / validCounter);
				//printf("StdDev %d: %f\n", i, stddev[i]);
			}
	
			// Mean of means and Mean of standard deviations
			for(i = 0; i < bands; i++){
				meanMeans += means[i];
				meanStddev += stddev[i];
			}
			meanMeans /= bands;
			meanStddev /= bands;

			hipMemcpy(d_images, images, sizeof(short)*arrSize, hipMemcpyHostToDevice);

			double *d_means;
			CUDA_MALLOC(d_means, bands, double);
			hipMemcpy(d_means, means, sizeof(double)*bands, hipMemcpyHostToDevice);

			double *d_stddev;
			CUDA_MALLOC(d_stddev, bands, double);
			hipMemcpy(d_stddev, stddev, sizeof(double)*bands, hipMemcpyHostToDevice);

			dim3 normBlocks(ceil(width/8.0),ceil(height/8.0),ceil(bands/8.0));
			dim3 normThreads(8,8,8);

			normImages16Kernel<<<normBlocks, normThreads>>>(d_images, d_means, meanMeans, d_stddev, meanStddev, width, height, bands, nullValue);
			hipHostFree(images);
			hipDeviceSynchronize();
			checkCUDAError("Error on Normalize Images Kernel");

			hipFree(d_means);
			hipFree(d_stddev);
		}
		// Compute Medians Band
		dim3 mediansBlocks(ceil(width/16.0), ceil(height/16.0), 1);
		dim3 mediansThreads(16,16,1);
		dcMediansBandInt16Kernel<<<mediansBlocks, mediansThreads>>>(d_images, d_medianc, width, height, bands, minValidCount, nullValue);
		hipDeviceSynchronize();
		checkCUDAError("Error on Medians Band Kernel");

		hipMemcpy(medianc, d_medianc, sizeof(short)*imgSize, hipMemcpyDeviceToHost);

		hipFree(d_images);
		hipFree(d_medianc);
	}
}

__global__ void applyMasks32Kernel(int *images, bool *cmasks, int width, int height, int bands, int nullValue){

	int x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y = threadIdx.y + (blockDim.y * blockIdx.y);
	int z = threadIdx.z + (blockDim.z * blockIdx.z);

	int idx = x + (y * width) + (z * height * width);

	if(x < width && y < height && z < bands){
		if(!cmasks[idx])
			images[idx] = nullValue;
	}

}
__global__ void normImages32Kernel(int *images, double *means, double meanMeans, double *stdDevs, double meanStdDevs, int width, int height, int  bands, int nullValue){

	int x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y = threadIdx.y + (blockDim.y * blockIdx.y);
	int z = threadIdx.z + (blockDim.z * blockIdx.z);

	int idx = x + (y * width) + (z * height * width);

	if(x < width && y < height && z < bands){
		if(images[idx] != nullValue)
			images[idx] = (((double)images[idx]-means[z])/stdDevs[z])*meanStdDevs+meanMeans;
	}
}

__global__ void dcMediansBandInt32Kernel(int *images, int *mediansBand, int width, int height, int bands, int minValidCount, int nullValue){

	int x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y = threadIdx.y + (blockDim.y * blockIdx.y);

	int idx = x + (y * width);
	int imgSize = width*height;

	int i, j, bandArrIdx;

	if(x < width && y < height){

		int validCounter = 0;
		int pixel[NBANDS];
		for(i = 0; i < bands; i++){
			bandArrIdx = idx + (imgSize*i);
			if(images[bandArrIdx] != nullValue){
				pixel[validCounter] = images[bandArrIdx];
				validCounter++;
			}
		}

		if(minValidCount <= validCounter){
			for(i = 0; i < validCounter-1; i++){
				for(j = i+1; j < validCounter; j++){
					if(pixel[i] > pixel[j]){
						int tmp = pixel[i];
						pixel[i] = pixel[j];
						pixel[j] = tmp;
					}
				}
			}
			mediansBand[idx] = MEDIAN(pixel, validCounter, int);
		}else{
			mediansBand[idx] = nullValue;
		}
	}
}

extern "C" {
	void dcMediansBandInt32(int *src_images, int *medianc, bool *cmask, int width, int height, int bands, bool normalized, int minValidCount, int nullValue){
	
		int i = 0, j = 0;
		int arrSize = width * height * bands;
		int imgSize = width * height;
		int xblocks, yblocks, zblocks;

		// Create images array on the CUDA device
		int *d_images;
		CUDA_MALLOC(d_images, arrSize, int);
		checkCUDAError("Error in CUDA Malloc for images");
		hipMemcpy(d_images, src_images, sizeof(int)*arrSize, hipMemcpyHostToDevice);
		checkCUDAError("Error in CUDA Memcpy for images");

		// Creates Cloud mask on the CUDA device
		bool *d_cmask;
		CUDA_MALLOC(d_cmask, arrSize, bool);
		checkCUDAError("Error in CUDA Malloc for cloud masks");	
		hipMemcpy(d_cmask, cmask, sizeof(bool)*arrSize, hipMemcpyHostToDevice);
		checkCUDAError("Error in CUDA Malloc for cloud masks");

		// Creates medians compound on the CUDA device
		int *d_medianc;
		CUDA_MALLOC(d_medianc, imgSize, int);
		checkCUDAError("Error in CUDA Malloc for Medians Band");

		// Apply Masks
		xblocks = ceil(width/8.0);
		yblocks = ceil(height/8.0);
		zblocks = ceil(bands/8.0);

		dim3 maskBlocks(xblocks, yblocks, zblocks);
		dim3 maskThreads(8,8,8);

		applyMasks32Kernel<<<maskBlocks, maskThreads>>>(d_images, d_cmask, width, height, bands, nullValue);
		hipDeviceSynchronize();
		checkCUDAError("Error in Apply Masks");

		hipFree(d_cmask);

		if(normalized){

			int *images;
			CUDA_MALLOC_HOST(images, arrSize, int);
			hipMemcpy(images, d_images, sizeof(int)*arrSize, hipMemcpyDeviceToHost);

			double means[bands];
			double stddev[bands];
			double meanMeans = 0.0;
			double meanStddev = 0.0;
			int validCounter, idx;

			// Mean for each band
			for(i = 0; i < bands; i++){
				validCounter = 0;
				means[i] = 0.0;
				for(j = 0; j < imgSize; j++){
					idx = j+(imgSize*i);
					if(images[idx] != nullValue){
						means[i] += images[idx];
						validCounter++;
					}
				}
				if(validCounter > 0)
					means[i] /= validCounter; 
				//printf("Means %d: %f\n", i, means[i]);
			}
	
			// Standard deviation for each band
			for(i = 0; i < bands; i++){
				validCounter = 0;
				stddev[i] = 0.0;
				for(j = 0; j < imgSize; j++){
					idx = j+(imgSize*i);
					if(images[idx] != nullValue){
						stddev[i] += pow((images[idx] - means[i]), 2);
						validCounter++;
					}
				}
				if(validCounter > 0)
					stddev[i] = sqrt(stddev[i] / validCounter);
				//printf("StdDev %d: %f\n", i, stddev[i]);
			}
	
			// Mean of means and Mean of standard deviations
			for(i = 0; i < bands; i++){
				meanMeans += means[i];
				meanStddev += stddev[i];
			}
			meanMeans /= bands;
			meanStddev /= bands;

			hipMemcpy(d_images, images, sizeof(int)*arrSize, hipMemcpyHostToDevice);

			double *d_means;
			CUDA_MALLOC(d_means, bands, double);
			hipMemcpy(d_means, means, sizeof(double)*bands, hipMemcpyHostToDevice);

			double *d_stddev;
			CUDA_MALLOC(d_stddev, bands, double);
			hipMemcpy(d_stddev, stddev, sizeof(double)*bands, hipMemcpyHostToDevice);

			dim3 normBlocks(ceil(width/8.0),ceil(height/8.0),ceil(bands/8.0));
			dim3 normThreads(8,8,8);

			normImages32Kernel<<<normBlocks, normThreads>>>(d_images, d_means, meanMeans, d_stddev, meanStddev, width, height, bands, nullValue);
			hipHostFree(images);
			hipDeviceSynchronize();
			checkCUDAError("Error on Normalize Images Kernel");

			hipFree(d_means);
			hipFree(d_stddev);
		}
		// Compute Medians Band
		dim3 mediansBlocks(ceil(width/16.0), ceil(height/16.0), 1);
		dim3 mediansThreads(16,16,1);
		dcMediansBandInt32Kernel<<<mediansBlocks, mediansThreads>>>(d_images, d_medianc, width, height, bands, minValidCount, nullValue);
		hipDeviceSynchronize();
		checkCUDAError("Error on Medians Band Kernel");

		hipMemcpy(medianc, d_medianc, sizeof(int)*imgSize, hipMemcpyDeviceToHost);

		hipFree(d_images);
		hipFree(d_medianc);
	}
}

__global__ void applyMasks64Kernel(double *images, bool *cmasks, int width, int height, int bands, double nullValue){

	int x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y = threadIdx.y + (blockDim.y * blockIdx.y);
	int z = threadIdx.z + (blockDim.z * blockIdx.z);

	int idx = x + (y * width) + (z * height * width);

	if(x < width && y < height && z < bands){
		if(!cmasks[idx])
			images[idx] = nullValue;
	}

}
__global__ void normImages64Kernel(double *images, double *means, double meanMeans, double *stdDevs, double meanStdDevs, int width, int height, int  bands, double nullValue){

	int x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y = threadIdx.y + (blockDim.y * blockIdx.y);
	int z = threadIdx.z + (blockDim.z * blockIdx.z);

	int idx = x + (y * width) + (z * height * width);

	if(x < width && y < height && z < bands){
		if(images[idx] != nullValue)
			images[idx] = (((double)images[idx]-means[z])/stdDevs[z])*meanStdDevs+meanMeans;
	}
}

__global__ void dcMediansBandFloat64Kernel(double *images, double *mediansBand, int width, int height, int bands, int minValidCount, double nullValue){

	int x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y = threadIdx.y + (blockDim.y * blockIdx.y);

	int idx = x + (y * width);
	int imgSize = width*height;

	int i, j, bandArrIdx;

	if(x < width && y < height){

		int validCounter = 0;
		double pixel[NBANDS];
		for(i = 0; i < bands; i++){
			bandArrIdx = idx + (imgSize*i);
			if(images[bandArrIdx] != nullValue){
				pixel[validCounter] = images[bandArrIdx];
				validCounter++;
			}
		}

		if(minValidCount <= validCounter){
			for(i = 0; i < validCounter-1; i++){
				for(j = i+1; j < validCounter; j++){
					if(pixel[i] > pixel[j]){
						double tmp = pixel[i];
						pixel[i] = pixel[j];
						pixel[j] = tmp;
					}
				}
			}
			mediansBand[idx] = MEDIAN(pixel, validCounter, double);
		}else{
			mediansBand[idx] = nullValue;
		}
	}
}

extern "C" {
	void dcMediansBandFloat64(double *src_images, double *medianc, bool *cmask, int width, int height, int bands, bool normalized, int minValidCount, double nullValue){
	
		int i = 0, j = 0;
		int arrSize = width * height * bands;
		int imgSize = width * height;
		int xblocks, yblocks, zblocks;

		// Create images array on the CUDA device
		double *d_images;
		CUDA_MALLOC(d_images, arrSize, double);
		checkCUDAError("Error in CUDA Malloc for images");
		hipMemcpy(d_images, src_images, sizeof(double)*arrSize, hipMemcpyHostToDevice);
		checkCUDAError("Error in CUDA Memcpy for images");

		// Creates Cloud mask on the CUDA device
		bool *d_cmask;
		CUDA_MALLOC(d_cmask, arrSize, bool);
		checkCUDAError("Error in CUDA Malloc for cloud masks");	
		hipMemcpy(d_cmask, cmask, sizeof(bool)*arrSize, hipMemcpyHostToDevice);
		checkCUDAError("Error in CUDA Malloc for cloud masks");

		// Creates medians compound on the CUDA device
		double *d_medianc;
		CUDA_MALLOC(d_medianc, imgSize, double);
		checkCUDAError("Error in CUDA Malloc for Medians Band");

		// Apply Masks
		xblocks = ceil(width/8.0);
		yblocks = ceil(height/8.0);
		zblocks = ceil(bands/8.0);

		dim3 maskBlocks(xblocks, yblocks, zblocks);
		dim3 maskThreads(8,8,8);

		applyMasks64Kernel<<<maskBlocks, maskThreads>>>(d_images, d_cmask, width, height, bands, nullValue);
		hipDeviceSynchronize();
		checkCUDAError("Error in Apply Masks");

		hipFree(d_cmask);

		if(normalized){

			double *images;
			CUDA_MALLOC_HOST(images, arrSize, double);
			hipMemcpy(images, d_images, sizeof(double)*arrSize, hipMemcpyDeviceToHost);

			double means[bands];
			double stddev[bands];
			double meanMeans = 0.0;
			double meanStddev = 0.0;
			int validCounter, idx;

			// Mean for each band
			for(i = 0; i < bands; i++){
				validCounter = 0;
				means[i] = 0.0;
				for(j = 0; j < imgSize; j++){
					idx = j+(imgSize*i);
					if(images[idx] != nullValue){
						means[i] += images[idx];
						validCounter++;
					}
				}
				if(validCounter > 0)
					means[i] /= validCounter; 
				//printf("Means %d: %f\n", i, means[i]);
			}
	
			// Standard deviation for each band
			for(i = 0; i < bands; i++){
				validCounter = 0;
				stddev[i] = 0.0;
				for(j = 0; j < imgSize; j++){
					idx = j+(imgSize*i);
					if(images[idx] != nullValue){
						stddev[i] += pow((images[idx] - means[i]), 2);
						validCounter++;
					}
				}
				if(validCounter > 0)
					stddev[i] = sqrt(stddev[i] / validCounter);
				//printf("StdDev %d: %f\n", i, stddev[i]);
			}
	
			// Mean of means and Mean of standard deviations
			for(i = 0; i < bands; i++){
				meanMeans += means[i];
				meanStddev += stddev[i];
			}
			meanMeans /= bands;
			meanStddev /= bands;

			hipMemcpy(d_images, images, sizeof(double)*arrSize, hipMemcpyHostToDevice);

			double *d_means;
			CUDA_MALLOC(d_means, bands, double);
			hipMemcpy(d_means, means, sizeof(double)*bands, hipMemcpyHostToDevice);

			double *d_stddev;
			CUDA_MALLOC(d_stddev, bands, double);
			hipMemcpy(d_stddev, stddev, sizeof(double)*bands, hipMemcpyHostToDevice);

			dim3 normBlocks(ceil(width/8.0),ceil(height/8.0),ceil(bands/8.0));
			dim3 normThreads(8,8,8);

			normImages64Kernel<<<normBlocks, normThreads>>>(d_images, d_means, meanMeans, d_stddev, meanStddev, width, height, bands, nullValue);
			hipHostFree(images);
			hipDeviceSynchronize();
			checkCUDAError("Error on Normalize Images Kernel");

			hipFree(d_means);
			hipFree(d_stddev);
		}
		// Compute Medians Band
		dim3 mediansBlocks(ceil(width/16.0), ceil(height/16.0), 1);
		dim3 mediansThreads(16,16,1);
		dcMediansBandFloat64Kernel<<<mediansBlocks, mediansThreads>>>(d_images, d_medianc, width, height, bands, minValidCount, nullValue);
		hipDeviceSynchronize();
		checkCUDAError("Error on Medians Band Kernel");

		hipMemcpy(medianc, d_medianc, sizeof(double)*imgSize, hipMemcpyDeviceToHost);

		hipFree(d_images);
		hipFree(d_medianc);
	}
}
