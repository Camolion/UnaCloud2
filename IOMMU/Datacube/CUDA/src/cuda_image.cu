#include "hip/hip_runtime.h"
#include "cuda_image.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define THREADS_PER_BLOCK 8

__global__ void negateImageKernel(int *image, int width, int height, int bands){

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;

	int idx = x + y * gridDim.x * blockDim.x + z * gridDim.x * blockDim.x * gridDim.y * blockDim.y;

	if( idx < (width * height * bands))
		image[idx] = 255 - image[idx];
}

/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

void printImage(int *image, int width, int height, int bands){

	int i, j, k;

	for(k = 0; k < bands; k++){
	
		printf("Band %d: \n", k);

		for(j = 0; j < height; j++){
		
			for(i = 0; i < width; i++){
	
				printf("%d\t", image[i + (width * j) + (width * height * k)]);
			}
			puts("");
		}
	
		printf("\n\n");
	}

}

extern "C" {
	void cudaNegateImage(int *image, int width, int height, int bands){

		int *d_image;
		int arrsize = width * height * bands;
		int xblocks = ceil((float)width/THREADS_PER_BLOCK);
		int yblocks = ceil((float)height/THREADS_PER_BLOCK);
		int zblocks = ceil((float)bands/THREADS_PER_BLOCK);

		hipMalloc((void **)&d_image, sizeof(int) * arrsize);

		checkCUDAError("Error in CUDA Malloc");
	
		hipMemcpy(d_image, image, sizeof(int) * arrsize, hipMemcpyHostToDevice);
	
		checkCUDAError("Error writing on Device");

		dim3 nblocks(xblocks, yblocks, zblocks);
		dim3 nthreads(THREADS_PER_BLOCK,THREADS_PER_BLOCK,THREADS_PER_BLOCK);

		printf("Number of blocks in x: %d\n", xblocks);
		printf("Number of blocks in y: %d\n", yblocks);
		printf("Number of blocks in z: %d\n", zblocks);
		printf("Number of threads per block: %d\n", THREADS_PER_BLOCK);
		printf("Total threads: %d\n", xblocks*yblocks*zblocks*THREADS_PER_BLOCK*THREADS_PER_BLOCK*THREADS_PER_BLOCK);
		printf("Total pixels: %d\n", arrsize);

		negateImageKernel<<<nblocks, nthreads>>>(d_image, width, height, bands);
		hipDeviceSynchronize();

		checkCUDAError("Error on kernel");

	//	printImage(image, width, height, bands);
		hipMemcpy(image, d_image, sizeof(int) * arrsize, hipMemcpyDeviceToHost);

		checkCUDAError("Error writing on Host");

	//	printImage(image, width, height, bands);
		hipFree(d_image);
	}
}
